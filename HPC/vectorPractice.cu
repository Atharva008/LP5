#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void arrAdd(int *x , int *y , int*z){
    int id = blockidx.x;
    z[id] =  x[id] + y[id];
}

int main(){

    int a[6] , b[6] , c[6];
    int *d , *e , *f;   
    printf("\n Enter elememts of first array \n")
    for(int i = 0 ; i < 6 ; i++){
        scanf("%d" , &a[i]);
    }

    printf("\n Enter elememts of second array \n")
    for(int i = 0 ; i < 6 ; i++){
        scanf("%d" , &b[i]);
    }

    hipMalloc((void**)&d , 6*sizeof(int));
    hipMalloc((void**)&e , 6*sizeof(int));
    hipMalloc((void**)&f , 6*sizeof(int));

    hipMemcpy(d , a , 6*sizeof(int) , cudaMemcpyHosttoDevice);
    hipMemcpy(e , b , 6*sizeof(int) , cudaMemcpyHosttoDevice);

    arrAdd<<<6,1>>>(d,e,f);

    hipMemcpy(c , f , 6*sizeof(int) , cudaMemcpyDevicetoHost);

    for(int i = 0 ; i < 6 ; i++){
        printf("%d" , c[i]);
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
    
}