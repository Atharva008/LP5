#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void arrAdd(int *x , int *y , int *z){
    int id = blockIdx.x;
    z[id] = x[id] + y[id];
}


int main(){
    int a[6] , b[6] ,  c[6];
    int d* , e* , f*;
    printf("\n Enter 6 elements in first array \n");
    for(int i = 0 ; i < 6 ; i++){
        scanf("%d" , &a[i]);
    }
    printf("\n Enter 6 elements in second array \n")
    for(int i = 0 ; i < 6 ; i++){
        scanf("%d" , &b[i]);
    }


    hipMalloc((void**)&d , 6*(sizeof(int)))
    hipMalloc((void**)&e , 6*(sizeof(int)))
    hipMalloc((void**)&f , 6*(sizeof(int)))

    hipMemcpy(d,a,6*(sizeof(int)),cudaMemcpyHosttoDevice)
    hipMemcpy(e,b,6*(sizeof(int)),cudaMemcpyHosttoDevice)

    arrAdd<<<6,1>>>(d,e,f);

    hipMemcpy(c,f,6*(sizeof(int)),cudaMemcpyDevicetoHost)
    printf("\n Sum of vectors is \n")
    for(int i = 0 ; i < 6 ; i++){
        printf("%d\t" , c[i]);
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

}